#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <exception>
#include <string>

void HandleCudaStatus(hipError_t status) {
	switch (status)
	{
		case hipSuccess: break;
		case hipErrorOutOfMemory: throw std::exception("Error in memory allocation");
		case hipErrorInvalidValue: throw std::exception("Invalid argument value");
		case hipErrorInvalidDevicePointer: throw std::exception("Invalid device pointer");
		case hipErrorInvalidMemcpyDirection: throw std::exception("Invalid copy dirrection");
		case hipErrorNotInitialized: throw std::exception("Error during initialization");
		case hipErrorPriorLaunchFailure: throw std::exception("Error in previous launch");
		case hipErrorInvalidHandle: throw std::exception("Invalid resource handler");
		default: throw std::exception(("Unrecognized cuda status: " + std::to_string(static_cast<int>(status))).c_str());
	}
}


__global__ void find_pi_number_using_integral(float* result)
{
	extern __shared__ float cache[];

	const float h = 1 / (float)(blockDim.x);
	const float x = h * threadIdx.x;
	cache[threadIdx.x] = sqrtf(1 - x * x) * h;

	__syncthreads();

	// reduction
	const size_t cacheIndex = threadIdx.x;
	size_t i = blockDim.x / 2;

	while (i != 0) {
		if (cacheIndex < i)
		{
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == i)
	{
		*result = 4.f * cache[0];
	}
}

int main()
{
	try
	{
		const size_t NUM_THREADS = 256;
		float *cudaPi;
		HandleCudaStatus(hipMalloc((void**)&cudaPi, sizeof(float)));

		find_pi_number_using_integral<<<1, NUM_THREADS, NUM_THREADS  * sizeof(float)>>>(cudaPi);
		HandleCudaStatus(hipGetLastError());

		float pi;
		HandleCudaStatus(hipMemcpy(&pi, cudaPi, sizeof(float), hipMemcpyDeviceToHost));

		std::cout << "Pi: " << pi << std::endl;
	}
	catch (std::exception& e)
	{
		std::cout << e.what() << std::endl;
	}
}
