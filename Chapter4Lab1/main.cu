#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <stdio.h>
#include <iostream>
#include <exception>
#include <string>

void HandleCudaStatus(hipError_t status) {
	switch (status)
	{
		case hipSuccess: break;
		case hipErrorOutOfMemory: throw std::exception("Error in memory allocation");
		case hipErrorInvalidValue: throw std::exception("Invalid argument value");
		case hipErrorInvalidDevicePointer: throw std::exception("Invalid device pointer");
		case hipErrorInvalidMemcpyDirection: throw std::exception("Invalid copy dirrection");
		case hipErrorNotInitialized: throw std::exception("Error during initialization");
		case hipErrorPriorLaunchFailure: throw std::exception("Error in previous launch");
		case hipErrorInvalidHandle: throw std::exception("Invalid resource handler");
		default: throw std::exception(("Unrecognized cuda status: " + std::to_string(static_cast<int>(status))).c_str());
	}
}


__device__ bool IsInCircle(hiprandState_t* state)
{
	float x = hiprand_uniform(state);
	float y = hiprand_uniform(state);
	return x * x + y * y <= 1.0f;
}


__global__ void CountPointsIntheCircle(unsigned int* result)
{

	//init hiprand
	hiprandState_t state;
	unsigned long long seed = (threadIdx.x + blockDim.x * blockIdx.x) + (threadIdx.y + blockDim.y * blockIdx.y) % 1000;
	hiprand_init(seed, 0, 0, &state);

	if (IsInCircle(&state))
	{
		atomicAdd(*result, 1);
	}
}

int main()
{
	try
	{
		const size_t NUM_THREADS = 512;
		unsigned int *cudaCounter;
		HandleCudaStatus(hipMalloc((void**)&cudaCounter, sizeof(unsigned int)));

		CountPointsIntheCircle<<<1, NUM_THREADS>>>(cudaCounter);
		HandleCudaStatus(hipGetLastError());

		unsigned int counter;
		HandleCudaStatus(hipMemcpy(&counter, cudaCounter, sizeof(unsigned int), hipMemcpyDeviceToHost));

		std::cout << "Pi: " << static_cast<float>(counter) / static_cast<float>(NUM_THREADS) << std::endl;
	}
	catch (std::exception& e)
	{
		std::cout << e.what() << std::endl;
	}
}
