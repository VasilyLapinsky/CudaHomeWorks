#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"

#include <iostream>

const size_t NUMBER_0F_POINTS = 1024 * 1024;
const size_t THREADS_PER_BLOCK = 1024;
const size_t BLOCKS_PER_GRID = std::min(size_t(32), (NUMBER_0F_POINTS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

hiprandGenerator_t CreateCuRandGenerator();
size_t CountPointsInTheCircle(float* randomX, float* randomY);

int main()
{
    // initialize clocks 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Allocate data
    float* randomX;
    float* randomY;
    hipMalloc((void**)&randomX, NUMBER_0F_POINTS * sizeof(float));
    hipMalloc((void**)&randomY, NUMBER_0F_POINTS * sizeof(float));

    // Generate random points
    hiprandGenerator_t gen = CreateCuRandGenerator();
    hiprandGenerateUniform(gen, randomX, NUMBER_0F_POINTS);
    hiprandGenerateUniform(gen, randomY, NUMBER_0F_POINTS);
    // claculates points in the circle
    size_t pointsInTheCircle = CountPointsInTheCircle(randomX, randomY);
    // calculates pi
    float pi = (4.f * static_cast<float>(pointsInTheCircle)) / static_cast<float>(NUMBER_0F_POINTS);

    std::cout << "Result pi: " << pi << '\n';

    // Free data in the end of the programm
    hipFree(randomX);
    hipFree(randomY);
    // calculate execution time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Elapsed time: " << elapsedTime << " ms\n";
    return 0;
}

hiprandGenerator_t CreateCuRandGenerator()
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    return gen;
}

__global__ void CalculatePointsIntheCircle(float* randomX, float* randomY, size_t* result)
{
    __shared__ size_t cache[THREADS_PER_BLOCK];
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    size_t cacheIndex = threadIdx.x;

    size_t temp = 0;
    float x, y;
    while (tid < NUMBER_0F_POINTS) {
        x = randomX[tid];
        y = randomY[tid];
        temp += sqrt(x * x + y * y) < 1.f ? 1 : 0;
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;
    __syncthreads();

    // reduction
    size_t i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
        {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
    {
        result[blockIdx.x] = cache[0];
    }
    __syncthreads();
}

size_t CountPointsInTheCircle(float* randomX, float* randomY)
{
    size_t* resultCountsDevice;
    hipMalloc((void**)&resultCountsDevice, BLOCKS_PER_GRID * sizeof(size_t));

    CalculatePointsIntheCircle << <BLOCKS_PER_GRID, THREADS_PER_BLOCK >> > (randomX, randomY, resultCountsDevice);

    size_t* resultCountsHost = new size_t[BLOCKS_PER_GRID];
    hipMemcpy(resultCountsHost, resultCountsDevice, BLOCKS_PER_GRID * sizeof(size_t), hipMemcpyDeviceToHost);

    size_t result = 0;
    for (size_t i = 0; i < BLOCKS_PER_GRID; ++i)
    {
        result += resultCountsHost[i];
    }


    hipFree(resultCountsDevice);
    delete[] resultCountsHost;

    return result;
}