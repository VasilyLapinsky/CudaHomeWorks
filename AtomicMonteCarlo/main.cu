#include <hip/hip_runtime.h>


#include "hiprand/hiprand_kernel.h"

#include <iostream>

const size_t WIDTH = 32;
const size_t HEIGHT = WIDTH;

//const size_t BLOCKS_WIDTH = std::min(size_t(32), (WIDTH*HEIGHT + WIDTH - 1) / WIDTH);
//const size_t BLOCKS_HEIGHT = BLOCKS_WIDTH;

__device__ float GenerateUniform(hiprandState_t *state)
{
    return hiprand_uniform(state);
}


__global__ void CalculatePointsIntheCircle(unsigned int* result)
{
    hiprandState_t state;
    unsigned long long seed = (threadIdx.x + blockDim.x * blockIdx.x) * threadIdx.y + blockDim.y * blockIdx.y;
    hiprand_init(seed, 0, 0, &state);

    float x = GenerateUniform(&state);
    float y = GenerateUniform(&state);

    if (x * x + y * y <= 1.f)
    {
        atomicAdd(result, 1u);
    }
}

int main()
{
    // initialize clocks 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    unsigned int countedPoints = 0;
    unsigned int *countedPointsDevice;
    hipMalloc((void**)&countedPointsDevice, sizeof(unsigned int));
    hipMemcpy(countedPointsDevice, &countedPoints, sizeof(unsigned int), hipMemcpyHostToDevice);

    dim3 blocks(1, 1, 1);
    dim3 threads(HEIGHT, WIDTH, 1);
    CalculatePointsIntheCircle<<<blocks, threads>>>(countedPointsDevice);

    hipMemcpy(&countedPoints, countedPointsDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);
    // calculates pi
    float pi = (4.f * static_cast<float>(countedPoints)) / static_cast<float>(HEIGHT*WIDTH);
    std::cout << "Result pi: " << pi << '\n';

    // calculate execution time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "Elapsed time: " << elapsedTime << " ms\n";
    return 0;
}