#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <exception>
#include <string>

void HandleCudaStatus(hipError_t status) {
	switch (status)
	{
		case hipSuccess: break;
		case hipErrorOutOfMemory: throw std::exception("Error in memory allocation");
		case hipErrorInvalidValue: throw std::exception("Invalid argument value");
		case hipErrorInvalidDevicePointer: throw std::exception("Invalid device pointer");
		case hipErrorInvalidMemcpyDirection: throw std::exception("Invalid copy dirrection");
		case hipErrorNotInitialized: throw std::exception("Error during initialization");
		case hipErrorPriorLaunchFailure: throw std::exception("Error in previous launch");
		case hipErrorInvalidHandle: throw std::exception("Invalid resource handler");
		default: throw std::exception(("Unrecognized cuda status: " + std::to_string(static_cast<int>(status))).c_str());
	}
}


__global__ void calculate_dzeta_fucntion(float* result, float s)
{
	extern __shared__ float cache[];

	cache[threadIdx.x] = 1.f / powf(float(threadIdx.x + 1), s);

	__syncthreads();

	// reduction
	const size_t cacheIndex = threadIdx.x;
	size_t i = blockDim.x / 2;

	while (i != 0) {
		if (cacheIndex < i)
		{
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == i)
	{
		*result = 4.f * cache[0];
	}
}

int main()
{
	try
	{
		const size_t NUM_THREADS = 512;
		const float S = 3.75;
		float *cudaDzeta;
		HandleCudaStatus(hipMalloc((void**)&cudaDzeta, sizeof(float)));

		calculate_dzeta_fucntion <<<1, NUM_THREADS, NUM_THREADS  * sizeof(float)>>>(cudaDzeta, S);
		HandleCudaStatus(hipGetLastError());

		float dzeta;
		HandleCudaStatus(hipMemcpy(&dzeta, cudaDzeta, sizeof(float), hipMemcpyDeviceToHost));

		std::cout << "Dzeta: " << dzeta << std::endl;
	}
	catch (std::exception& e)
	{
		std::cout << e.what() << std::endl;
	}
}
