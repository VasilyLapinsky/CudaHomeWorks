#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <exception>
#include <string>
#include <chrono>
#include <type_traits>

const size_t NUM_THREADS = 512;

void HandleCudaStatus(hipError_t status) {
	switch (status)
	{
	case hipSuccess: break;
	case hipErrorOutOfMemory: throw std::exception("Error in memory allocation");
	case hipErrorInvalidValue: throw std::exception("Invalid argument value");
	case hipErrorInvalidDevicePointer: throw std::exception("Invalid device pointer");
	case hipErrorInvalidMemcpyDirection: throw std::exception("Invalid copy dirrection");
	case hipErrorNotInitialized: throw std::exception("Error during initialization");
	case hipErrorPriorLaunchFailure: throw std::exception("Error in previous launch");
	case hipErrorInvalidHandle: throw std::exception("Invalid resource handler");
	default: throw std::exception(("Unrecognized cuda status: " + std::to_string(static_cast<int>(status))).c_str());
	}
}

template<typename T>
__global__ void fill(T* matrix, int size, T val) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = idx; i < size; i += gridDim.x * blockDim.x) {
		matrix[i] = val;
	}
}

__global__ void VectorMult(float* left, float* right, float* result, int size)
{
	extern __shared__ float resultcache[];

	const size_t cacheIndex = threadIdx.x;
	resultcache[cacheIndex] = 0;

	int tid = threadIdx.x;
	while (tid < size)
	{
		resultcache[cacheIndex] += left[tid] * right[tid];
		tid += gridDim.x * blockDim.x;
	}

	__syncthreads();

	// reduction
	size_t i = blockDim.x / 2;

	while (i != 0) {
		if (cacheIndex < i)
		{
			resultcache[cacheIndex] += resultcache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0)
	{
		*result = resultcache[0];
	}
}

int main()
{
	try
	{
		const size_t size = 30000;

		const auto startCalculation = std::chrono::system_clock::now();

		float* vector;
		HandleCudaStatus(hipMalloc((void**)&vector, size * sizeof(float)));
		fill<<<1, NUM_THREADS >>>(vector, size, 5.f);
		HandleCudaStatus(hipGetLastError());

		float* result;
		HandleCudaStatus(hipMalloc((void**)&result, sizeof(float)));
		VectorMult<<<1, NUM_THREADS, NUM_THREADS * sizeof(float)>>>(vector, vector, result, size);
		HandleCudaStatus(hipGetLastError());

		float resultCpu;
		HandleCudaStatus(hipMemcpy((void*)&resultCpu, result, sizeof(float), hipMemcpyDeviceToHost));

		HandleCudaStatus(hipFree(vector));
		HandleCudaStatus(hipFree(result));

		std::cout << "result: " << sqrt(resultCpu) << '\n';
		auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::system_clock::now() - startCalculation);
		std::cout << "Duration: " << duration.count() << " milliseconds\n";
	}
	catch (std::exception& e)
	{
		std::cout << e.what() << std::endl;
	}
}
